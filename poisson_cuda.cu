#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdbool.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <math.h>

#include "utils.h"

#include "cuda_worker.cuh"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

#define PRECISION double
#define BLOCK_SIZE 4

uint64_t vram_capacity;

/**
 * poisson.c
 * Implementation of a Poisson solver with Dirichlet boundary conditions.
 *
 * This template handles the basic program launch, argument parsing, and memory
 * allocation required to implement the solver *at its most basic level*. You
 * will likely need to allocate more memory, add threading support, account for
 * cache locality, etc...
 *
 * BUILDING:
 * gcc -o poisson poisson.c -lpthread
 *
 * [note: linking pthread isn't strictly needed until you add your
 *        multithreading code]
 *
 * TODO:
 * 1 - Read through this example, understand what it does and what it gives you
 *     to work with.
 * 2 - Implement the basic algorithm and get a correct output.
 * 3 - Add a timer to track how long your execution takes.
 * 4 - Profile your solution and identify weaknesses.
 * 5 - Improve it!
 * 6 - Remember that this is now *your* code and *you* should modify it however
 *     needed to solve the assignment.
 *
 * See the lab notes for a guide on profiling and an introduction to
 * multithreading (see also threads.c which is reference by the lab notes).
 */

extern char* optarg;

// Global flag
// Set to true when operating in debug mode to enable verbose logging
static bool debug = false;

// Statics
const double top_boundary_cond = -1; // V The top dirlec boundary condition
const double bottom_boundary_cond = 1; // V The bottom dirlec boundary condition

void apply_const_boundary(int N, double* next) {
    for (int j = 0; j < N; j++) {
        for (int i = 0; i < N; i++) {
            idx(next, N, 0, j, i) = top_boundary_cond;
            idx(next, N, N - 1, j, i) = bottom_boundary_cond;
        }
    }
}

/**
 * @brief Solve Poissons equation for a given cube with Dirichlet boundary
 * conditions on all sides.
 *
 * @param N             The edge length of the cube. n^3 number of elements.
 * @param source        Pointer to the source term cube, a.k.a. forcing function.
 * @param iterations    Number of iterations to perform.
 * @param threads       Number of threads to use for solving.
 * @param delta         Grid spacing.
 * @return double*      Solution to Poissons equation.  Caller must free.
 */
double* poisson_mixed(int N, double* source, int iterations, float delta) {
    if (debug) {
        printf("Starting solver with:\n"
            "n = %i\n"
            "iterations = %i\n"
            "delta = %f\n",
            N, iterations, delta);
    }

    // Allocate memory for the solution on the host
    double* curr = (double*)calloc(N * N * N, sizeof(double));
    double* next = (double*)calloc(N * N * N, sizeof(double));
    // Ensure we haven't run out of memory
    if (curr == NULL || next == NULL) {
        fprintf(stderr, "Error: ran out of memory when trying to allocate %i sized cube\n", N);
        exit(EXIT_FAILURE);
    }

    // Apply constant boundary
    apply_const_boundary(N, next);

    uint64_t used_memory = N * N * N * sizeof(double) * 3;
    if (debug) {
        printf("Predicting memory use of %lu\n", used_memory);
    }

    uint64_t batches = (uint16_t)ceil((float)used_memory / (float)vram_capacity);
    if (debug) {
        printf("Identified need for %lu batches per iteration\n", batches);
    }

    // Allocate device memory
    uint64_t batch_size = N * N * N * sizeof(double) / batches; 
    double *d_source, *d_curr, *d_next;
    hipError_t ex;
    
    ex = hipMalloc((void**)&d_source, batch_size);
    if (ex != 0) {
        fprintf(stderr, "Error: ran out of memory when trying to allocate %i sized cube on GPU, error code %i\n", N, ex);
        exit(EXIT_FAILURE);
    }
    ex = hipMalloc((void**)&d_curr, batch_size);
    if (ex != 0) {
        fprintf(stderr, "Error: ran out of memory when trying to allocate %i sized cube on GPU, error code %i\n", N, ex);
        exit(EXIT_FAILURE);
    }
    ex = hipMalloc((void**)&d_next, batch_size);
    if (ex != 0) {
        fprintf(stderr, "Error: ran out of memory when trying to allocate %i sized cube on GPU, error code %i\n", N, ex);
        exit(EXIT_FAILURE);
    }

    // Copy data to device
    

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                  (N + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                  (N + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Main iteration loop
    for (int iter = 0; iter < iterations; iter++) {
        for (int batch_num = 0; batch_num < batches; batch_num++)
        {
            /// SOMETHING NEEDED HERE...
            // hipMemcpy(d_source, &(source[batch_num * batch_size]), batch_size, hipMemcpyHostToDevice);
            // hipMemcpy(d_curr, next, N * N * N * sizeof(double), hipMemcpyHostToDevice);
        }
        poisson_slice<<<numBlocks, threadsPerBlock>>>(N, d_source, d_curr, d_next, delta);

        // Swap pointers
        double* temp = d_curr;
        d_curr = d_next;
        d_next = temp;
    }

    // Copy the result back to the host
    hipMemcpy(curr, d_curr, N * N * N * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_source);
    hipFree(d_curr);
    hipFree(d_next);
    free(next); // free the next buffer as it's no longer needed

    if (debug) {
        printf("Finished solving.\n");
    }

    return curr; // Return the result
}

int main(int argc, char** argv) {
    // Default settings for solver
    int iterations = 300;
    int n = 5;
    int threads = 3;
    float delta = 1;
    int x = -1;
    int y = -1;
    int z = -1;
    double amplitude = 1.0;

    int opt;

    system("nvidia-smi --query-gpu=memory.total --format=csv,noheader,nounits");

    char buff[16];
    FILE *f = fopen("vram.txt", "r");
    fgets(buff, 16, f);
    vram_capacity = atoi(buff); 
    vram_capacity = vram_capacity * 1024 * 1024 * 4 / 5; // Convert from MB to B
    printf("vram_capacity: %lu\n", vram_capacity);

    // parse the command line arguments
    while ((opt = getopt(argc, argv, "h:n:i:x:y:z:a:t:d:")) != -1) {
        switch (opt) {
        case 'h':
            printf("Usage: poisson [-n size] [-x source x-poisition] [-y source y-position] [-z source z-position] [-a source amplitude] [-i iterations] [-t threads] [-d] (for debug mode)\n");
            return EXIT_SUCCESS;
        case 'n':
            n = atoi(optarg);
            break;
        case 'i':
            iterations = atoi(optarg);
            break;
        case 'x':
            x = atoi(optarg);
            break;
        case 'y':
            y = atoi(optarg);
            break;
        case 'z':
            z = atoi(optarg);
            break;
        case 'a':
            amplitude = atof(optarg);
            break;
        case 't':
            threads = atoi(optarg);
            break;
        case 'd':
            debug = true;
            break;
        default:
            fprintf(stderr, "Usage: poisson [-n size] [-x source x-poisition] [-y source y-position] [-z source z-position] [-a source amplitude]  [-i iterations] [-t threads] [-d] (for debug mode)\n");
            exit(EXIT_FAILURE);
        }
    }

    // Ensure we have an odd sized cube
    if (n % 2 == 0) {
        fprintf(stderr, "Error: n should be an odd number!\n");
        return EXIT_FAILURE;
    }

    // Create a source term with a single point in the centre
    double* source = (double*)calloc(n * n * n, sizeof(double));
    if (source == NULL) {
        fprintf(stderr, "Error: failed to allocated source term (n=%i)\n", n);
        return EXIT_FAILURE;
    }

    // Default x,y, z
    if (x < 0 || x > n - 1)
        x = n / 2;
    if (y < 0 || y > n - 1)
        y = n / 2;
    if (z < 0 || z > n - 1)
        z = n / 2;

    source[(z * n + y) * n + x] = amplitude;

    // Calculate the resulting field with mixed boundary conditions
    double* result = poisson_mixed(n, source, iterations, delta);

    // Print out the middle slice of the cube for validation
    if (debug) {
        printf("--MIDDLE--\n");
    }
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%0.5f ", result[((n / 2) * n + y) * n + x]);
        }
        printf("\n");
    }

    free(source);
    free(result);
    
    return EXIT_SUCCESS;
}
