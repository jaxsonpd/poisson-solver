#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <stdio.h>

#include "utils.h"

#include "cuda_worker.cuh"

constexpr int BLOCK_SIZE = 8;
// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

// template <int BLOCK_SIZE>
__global__ void apply_von_neuman_boundary_slice(int N, double *source, double *curr, double *next, float delta) {
    // Calculate 3D indices for the current thread
    int block_k = blockIdx.z * BLOCK_SIZE + threadIdx.z;
    int block_j = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int block_i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (block_k >= N || block_j >= N || block_i >= N) return; // Ensure within bounds

    // Apply boundary conditions (as per your logic)
    // Example for boundaries as in your original function
    if (block_j == 0 && block_i == 0) {
        idx(next, N, block_k, 0, 0) = (2 * idx(curr, N, block_k, 0, 0 + 1)
            + 2 * idx(curr, N, block_k, 0 + 1, 0)
            + idx(curr, N, block_k + 1, 0, 0) + idx(curr, N, block_k - 1, 0, 0)
            - delta * delta * idx(source, N, block_k, 0, 0)) / 6;
    }

    if (block_j == N && block_j == N) {
        idx(next, N, block_k, N - 1, N - 1) = (2 * idx(curr, N, block_k, N - 1, N - 1 - 1)
            + 2 * idx(curr, N, block_k, N - 1 - 1, N - 1)
            + idx(curr, N, block_k + 1, N - 1, N - 1) + idx(curr, N, block_k - 1, N - 1, N - 1)
            - delta * delta * idx(source, N, block_k, N - 1, N - 1)) / 6;
    }

    if (block_j == N && block_i == 0) {
        idx(next, N, block_k, N - 1, 0) = (2 * idx(curr, N, block_k, N - 1, 0 + 1)
            + 2 * idx(curr, N, block_k, N - 1 - 1, 0)
            + idx(curr, N, block_k + 1, N - 1, 0) + idx(curr, N, block_k - 1, N - 1, 0)
            - delta * delta * idx(source, N, block_k, N - 1, 0)) / 6;
    }

    if (block_j == 0 && block_j == N) {
        idx(next, N, block_k, 0, N - 1) = (2 * idx(curr, N, block_k, 0, N - 1 - 1)
            + 2 * idx(curr, N, block_k, 0 + 1, N - 1)
            + idx(curr, N, block_k + 1, 0, N - 1) + idx(curr, N, block_k - 1, 0, N - 1)
            - delta * delta * idx(source, N, block_k, 0, N - 1)) / 6;
    }

    if (block_i == 0) {
        for (int j = block_j + 1; j < block_j - 1; j++) {
            idx(next, N, block_k, j, 0) = (2 * idx(curr, N, block_k, j, 0 + 1)
                + idx(curr, N, block_k, j + 1, 0) + idx(curr, N, block_k, j - 1, 0)
                + idx(curr, N, block_k + 1, j, 0) + idx(curr, N, block_k - 1, j, 0)
                - delta * delta * idx(source, N, block_k, j, 0)) / 6;
        }
    }

    if (block_i == N) {
        for (int j = block_j + 1; j < block_j - 1; j++) {
            idx(next, N, block_k, j, N - 1) = (2 * idx(curr, N, block_k, j, N - 1 - 1)
                + idx(curr, N, block_k, j + 1, N - 1) + idx(curr, N, block_k, j - 1, N - 1)
                + idx(curr, N, block_k + 1, j, N - 1) + idx(curr, N, block_k - 1, j, N - 1)
                - delta * delta * idx(source, N, block_k, j, N - 1)) / 6;
        }
    }

    if (block_j == 0) {
        for (int i = block_i + 1; i < block_i - 1; i++) {
            idx(next, N, block_k, 0, i) = (idx(curr, N, block_k, 0, i + 1) + idx(curr, N, block_k, 0, i - 1)
                + 2 * idx(curr, N, block_k, 0 + 1, i)
                + idx(curr, N, block_k + 1, 0, i) + idx(curr, N, block_k - 1, 0, i)
                - delta * delta * idx(source, N, block_k, 0, i)) / 6;
        }
    }

    if (block_j == N) {
        for (int i = block_i + 1; i < block_i - 1; i++) {
            idx(next, N, block_k, N - 1, i) = (idx(curr, N, block_k, N - 1, i + 1) + idx(curr, N, block_k, N - 1, i - 1)
                + 2 * idx(curr, N, block_k, N - 1 - 1, i)
                + idx(curr, N, block_k + 1, N - 1, i) + idx(curr, N, block_k - 1, N - 1, i)
                - delta * delta * idx(source, N, block_k, N - 1, i)) / 6;
        }
    }

    // Other boundary conditions go here
}

// template <int BLOCK_SIZE>
__global__ void poisson_iteration_inner_slice(int N, double *source, double *curr, double *next, float delta) {
    // Calculate 3D indices for the current thread
    int k = blockIdx.z * BLOCK_SIZE + threadIdx.z;
    int j = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (k >= N || j >= N || i >= N) return; // Ensure within bounds

    // Perform the inner iteration logic
    idx(next, N, k, j, i) = (idx(curr, N, k, j, i + 1) + idx(curr, N, k, j, i - 1)
                            + idx(curr, N, k, j + 1, i) + idx(curr, N, k, j - 1, i)
                            + idx(curr, N, k + 1, j, i) + idx(curr, N, k - 1, j, i)
                            - delta * delta * idx(source, N, k, j, i)) / 6;
}

// void* worker_thread(void* pargs) {
//     workerThread_t* worker_info = (workerThread_t*)pargs;
//     int N = worker_info->N;

//     // Device memory allocation
//     double *d_source, *d_curr, *d_next;
//     hipError_t err;

//     err = hipMalloc((void**)&d_source, N * N * N * sizeof(double));
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error (malloc d_source): %s\n", hipGetErrorString(err));
//         return NULL;
//     }
//     err = hipMalloc((void**)&d_curr, N * N * N * sizeof(double));
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error (malloc d_curr): %s\n", hipGetErrorString(err));
//         hipFree(d_source);
//         return NULL;
//     }
//     err = hipMalloc((void**)&d_next, N * N * N * sizeof(double));
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error (malloc d_next): %s\n", hipGetErrorString(err));
//         hipFree(d_source);
//         hipFree(d_curr);
//         return NULL;
//     }

//     // Copy data to device
//     err = hipMemcpy(d_source, worker_info->source, N * N * N * sizeof(double), hipMemcpyHostToDevice);
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error (memcpy to d_source): %s\n", hipGetErrorString(err));
//         hipFree(d_source);
//         hipFree(d_curr);
//         hipFree(d_next);
//         return NULL;
//     }

//     // Define kernel launch configuration
//     dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
//     dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

//     // Create a CUDA stream
//     hipStream_t stream;
//     hipStreamCreate(&stream);

//     for (int n = 0; n < worker_info->iterations; n++) {
//         // Call the boundary condition kernel
//         apply_von_neuman_boundary_slice<BLOCK_SIZE><<<numBlocks, threadsPerBlock, 0, stream>>>(N, d_source, d_curr, d_next, worker_info->delta);

//         // Call the inner iteration kernel
//         poisson_iteration_inner_slice<BLOCK_SIZE><<<numBlocks, threadsPerBlock, 0, stream>>>(N, d_source, d_curr, d_next, worker_info->delta);

//         // Check for errors after kernel launches
//         err = hipGetLastError();
//         if (err != hipSuccess) {
//             fprintf(stderr, "CUDA error (kernel launch): %s\n", hipGetErrorString(err));
//             break;
//         }

//         // Synchronize before copying data back
//         hipStreamSynchronize(stream);

//         // Memory copy back to host if needed
//         err = hipMemcpy(worker_info->curr, d_next, N * N * N * sizeof(double), hipMemcpyDeviceToHost);
//         if (err != hipSuccess) {
//             fprintf(stderr, "CUDA error (memcpy to host): %s\n", hipGetErrorString(err));
//             break;
//         }
//     }

//     // Clean up
//     hipStreamDestroy(stream);
//     hipFree(d_source);
//     hipFree(d_curr);
//     hipFree(d_next);

//     return NULL;
// }
